#include "hip/hip_runtime.h"
# include <assert.h>
#include <stdio.h>
#include "freshman.h"


__global__ void sum_matrix_gpu(float *a, float *b, float *c, uint const w, uint const h)
{
    uint x = threadIdx.x + blockIdx.x * blockDim.x;
    uint y = threadIdx.y + blockIdx.y * blockDim.y; 
    uint idx = w * y + x;
    c[idx] = a[idx] + b[idx];
}

void sum_matrix_cpu(float *a, float *b, float *c, uint const w, uint const h)
{
    for (int y = 0; y < h; y++)
    {
        for (int x = 0; x < w; x++)
        {
            uint idx = w * y + x;
            c[idx] = a[idx] + b[idx];
        }
    }
}


int main()
{
    uint w = 1000;
    uint h = 1000;
    uint num_ele = w * h;
    float *a_cpu = new float[num_ele];
    float *b_cpu = new float[num_ele];
    float *c_cpu = new float[num_ele];

    initialData(a_cpu, num_ele);
    initialData(b_cpu, num_ele);
    initialData(c_cpu, num_ele);

    TIMEIT(sum_matrix_cpu(a_cpu, b_cpu, c_cpu, w, h), "sum_matrix_cpu");

    float *a_gpu;
    float *b_gpu;
    float *c_gpu;
    float c_from_gpu[num_ele];
    hipMalloc((float**)&a_gpu, num_ele * sizeof(float));
    hipMalloc((float**)&b_gpu, num_ele * sizeof(float));
    hipMalloc((float**)&c_gpu, num_ele * sizeof(float));
    
    hipMemcpy(a_gpu, a_cpu, num_ele * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, num_ele * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_gpu, c_cpu, num_ele * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(100, 100);
    dim3 grid((w - 1) / block.x + 1, (h - 1) / block.y + 1);

    CUDATIMEIT(sum_matrix_gpu, grid, block, "sum_matrix_gpu", a_gpu, b_gpu, c_gpu, w, h);
    hipMemcpy(c_from_gpu, c_gpu, num_ele * sizeof(float), hipMemcpyDeviceToHost);

    checkResult(c_from_gpu, c_cpu, num_ele);
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
}
